
#include <hip/hip_runtime.h>
// Copyright 2022 Aqrose Technology, Ltd.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef USE_CUDA

#include "example.hpp"
#include <cstdio>

__global__ void kernel() {
  printf("kernel id = %d\n", blockIdx.x * blockDim.x + threadIdx.x);
}

int launch() {
  kernel<<<2, 3>>>();
  return 0;
}

#endif
